#include <torch/extension.h>
#include <hipsparse.h>
#include <hipblas.h>
#include <vector>

torch::Tensor cusparse_coo_multiply(
    torch::Tensor cooRowInd,
    torch::Tensor cooColInd,
    torch::Tensor cooVal,
    torch::Tensor x,
    int A_num_rows,
    int A_num_cols
   ) {
    AT_ASSERTM(cooVal.is_cuda(), "cooVal must be a CUDA tensor");
    AT_ASSERTM(cooRowInd.is_cuda(), "cooRowInd must be a CUDA tensor");
    AT_ASSERTM(cooColInd.is_cuda(), "cooColInd must be a CUDA tensor");
    AT_ASSERTM(x.is_cuda(), "x must be a CUDA tensor");
    auto options = cooVal.options();
    auto y = torch::zeros({A_num_rows}, options);

    hipsparseHandle_t handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnVecDescr_t vecX, vecY;
    double alpha = 1.0, beta = 0.0;
    void* dBuffer = NULL;
    size_t bufferSize = 0;

    hipsparseCreate(&handle);

    // Create sparse matrix A in COO format
    hipsparseCreateCoo(&matA,
                      A_num_rows, A_num_cols, cooVal.numel(),
                      cooRowInd.data_ptr<int>(), cooColInd.data_ptr<int>(), cooVal.data_ptr<double>(),
                      HIPSPARSE_INDEX_32I, // Index base
                      HIPSPARSE_INDEX_BASE_ZERO, // Index base
                      HIP_R_64F); // Data type

    // Create dense vector x and y
    hipsparseCreateDnVec(&vecX, x.numel(), x.data_ptr<double>(), HIP_R_64F);
    hipsparseCreateDnVec(&vecY, y.numel(), y.data_ptr<double>(), HIP_R_64F);

    // Compute buffer size and allocate buffer
    hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                            &alpha, matA, vecX, &beta, vecY, HIP_R_64F,
                            HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize);
    hipMalloc(&dBuffer, bufferSize);

    // Perform matrix-vector multiplication
    hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                 &alpha, matA, vecX, &beta, vecY, HIP_R_64F,
                 HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer);

    // Cleanup
    hipsparseDestroySpMat(matA);
    hipsparseDestroyDnVec(vecX);
    hipsparseDestroyDnVec(vecY);
    hipFree(dBuffer);
    hipsparseDestroy(handle);

    return y;
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor> cusparse_csr_ichol(
    torch::Tensor csrRowInd,
    torch::Tensor csrColInd,
    torch::Tensor csrVal,
    int A_num_rows,
    int A_num_cols,
    int nnz
   ) {
    AT_ASSERTM(csrVal.is_cuda(), "cooVal must be a CUDA tensor");
    AT_ASSERTM(csrRowInd.is_cuda(), "cooRowInd must be a CUDA tensor");
    AT_ASSERTM(csrColInd.is_cuda(), "cooColInd must be a CUDA tensor");
    auto csrVal1 = csrVal.clone();
    auto csrRowInd1 = csrRowInd.clone();
    auto csrColInd1 = csrColInd.clone();

    auto options = csrVal.options();
    int num_offsets = csrRowInd.numel();
    int m = A_num_rows;

    // Create cuSPARSE matrix descriptor for L
    hipsparseFillMode_t   fill_lower    = HIPSPARSE_FILL_MODE_LOWER;
    hipsparseDiagType_t   diag_non_unit = HIPSPARSE_DIAG_TYPE_NON_UNIT;

    hipsparseIndexBase_t  baseIdx = HIPSPARSE_INDEX_BASE_ZERO;
    hipsparseSpMatDescr_t matL;
    hipsparseCreateCsr(&matL, A_num_rows, A_num_cols, nnz, csrRowInd1.data_ptr<int>(),
                                      csrColInd1.data_ptr<int>(), csrVal1.data_ptr<double>(),
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      baseIdx, HIP_R_64F);
    hipsparseSpMatSetAttribute(matL,
                              HIPSPARSE_SPMAT_FILL_MODE,
                              &fill_lower, sizeof(fill_lower));

    hipsparseSpMatSetAttribute(matL,
                              HIPSPARSE_SPMAT_DIAG_TYPE,
                              &diag_non_unit,
                              sizeof(diag_non_unit));
    hipsparseHandle_t cusparseHandle = NULL;
    hipsparseCreate(&cusparseHandle);

    // Incomplete Cholesky factorization
    hipsparseMatDescr_t descrM;
    csric02Info_t      infoM        = NULL;
    int                bufferSizeIC = 0;
    void*              d_bufferIC;
    hipsparseCreateMatDescr(&descrM);
    hipsparseSetMatIndexBase(descrM, baseIdx);
    hipsparseSetMatType(descrM, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatFillMode(descrM, HIPSPARSE_FILL_MODE_LOWER);
    hipsparseSetMatDiagType(descrM, HIPSPARSE_DIAG_TYPE_NON_UNIT);
    //creates and initializes the solve and analysis structure of incomplete Cholesky to default values
    hipsparseCreateCsric02Info(&infoM);
    //returns size of buffer used in computing the incomplete-Cholesky factorization
    hipsparseDcsric02_bufferSize(
                        cusparseHandle, m, nnz, descrM, csrVal1.data_ptr<double>(),
                        csrRowInd1.data_ptr<int>(), csrColInd1.data_ptr<int>(), infoM, &bufferSizeIC);
    //performs the analysis phase of the incomplete-Cholesky factorization
    hipMalloc(&d_bufferIC, bufferSizeIC);
    hipsparseDcsric02_analysis(
                        cusparseHandle, m, nnz, descrM, csrVal1.data_ptr<double>(),
                        csrRowInd1.data_ptr<int>(), csrColInd1.data_ptr<int>(), infoM,
                        HIPSPARSE_SOLVE_POLICY_NO_LEVEL, d_bufferIC);
    int structural_zero;
    //know where the structural zero is.
    hipsparseXcsric02_zeroPivot(cusparseHandle, infoM,
                                               &structural_zero);
    hipsparseDcsric02(
                        cusparseHandle, m, nnz, descrM, csrVal1.data_ptr<double>(),
                        csrRowInd1.data_ptr<int>(), csrColInd1.data_ptr<int>(), infoM,
                        HIPSPARSE_SOLVE_POLICY_NO_LEVEL, d_bufferIC);
    // Find numerical zero
    int numerical_zero;
    hipsparseXcsric02_zeroPivot(cusparseHandle, infoM,
                                               &numerical_zero);

    hipsparseDestroyCsric02Info(infoM);
    hipsparseDestroyMatDescr(descrM);
    hipFree(d_bufferIC);

    hipsparseDestroy(cusparseHandle);

    auto result = std::make_tuple(csrRowInd1, csrColInd1, csrVal1);
    return result;
}


PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("coo_multiply", &cusparse_coo_multiply, "Sparse matrix-vector multiplication (COO format) using cuSparseSpMV");
  m.def("csr_ichol", &cusparse_csr_ichol, "Incomplete Cholesky factorization (CSR format) using cuSparse");
}
